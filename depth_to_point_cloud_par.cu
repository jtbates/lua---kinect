#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "preprocess.h"

#include <iostream>

#define HEIGHT 480
#define WIDTH 640

// The maximum observable depth, in meters.
#define MAX_DEPTH 10

// Here are all the camera parameters
#define FX_RGB 5.1930334103339817e+02
#define FY_RGB 5.1816401430246583e+02
#define CX_RGB 3.2850951551345941e+02
#define CY_RGB 2.5282555217253503e+02

// Distortion coefficients
#define K1_RGB 2.5785516449232132e-01
#define K2_RGB -9.1141470196267182e-01
#define P1_RGB 3.0173013316440469e-04
#define P2_RGB 2.5422024034001231e-03
#define K3_RGB 1.1823504884394158e+00

// Depth camera parameters
#define FX_D 5.7616540758591043e+02
#define FY_D 5.7375619782082447e+02
#define CX_D 3.2442516903961865e+02
#define CY_D 2.3584766381177013e+02

#define K1_D  -1.3708537316819339e-01
#define K2_D 7.2482751812234414e-01
#define P1_D 8.0826809257389550e-04
#define P2_D 3.4151576458975323e-03
#define K3_D -1.4621396186358457e+00

// Inverse Rotation matrix in column major order.
#define R1 0.999985794494467
#define R2 -0.003429138557773
#define R3 0.00408066391266
#define R4 0.003420377768765
#define R5 0.999991835033557
#define R6 0.002151948451469
#define R7 -0.004088009930192
#define R8 -0.002137960469802
#define R9 0.999989358593300

// Translation vector
#define T1 -2.2142187053089738e-02
#define T2 1.4391632009665779e-04
#define T3 7.9356552371601212e-03

// Constants for undoing the depth nonlinearity.
#define DN_W 0.3513e3
#define DN_B 1.0925e3

// Args:
//   depth_abs - the absolute depth from the kinect.
//   depth_proj - the projected depth.
__global__ void depth_to_point_cloud(const float* depth_abs, int* res_x, int* res_y,
                                     const int N) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= N) {
	  return;
	}
	
  // Figure out the current XY coordinates. These coordinates are actually
	// 1-indexed, not zero-indexed.
  int x = idx % WIDTH + 1;
	int y = static_cast<int>(floorf(idx / WIDTH) + 1);
	
  // ****************************************
	//   PROJECT THE DEPTH TO 3D WORLD POINTS
	// ****************************************
  float x_world = (x - CX_D) * depth_abs[idx] / FX_D;
	float y_world = (y - CY_D) * depth_abs[idx] / FY_D;
	float z_world = depth_abs[idx];
		
	// *******************************************
	//   Next, Rotate and translate the 3D points
	// *******************************************
	// R * [X; Y; Z] + T
	
	float x_tmp = x_world;
	float y_tmp = y_world;
	float z_tmp = z_world;
  x_world = (R1 * x_tmp) + (R2 * y_tmp) + (R3 * z_tmp) + T1;
	y_world = (R4 * x_tmp) + (R5 * y_tmp) + (R6 * z_tmp) + T2;
	z_world = (R7 * x_tmp) + (R8 * y_tmp) + (R9 * z_tmp) + T3;
	
	// *******************************************
	//   Project into the RGB coordinate frame.
	// *******************************************	
	float x_proj = x_world * FX_RGB / z_world + CX_RGB;
	float y_proj = y_world * FY_RGB / z_world + CY_RGB;
	
	// ************************************************
	// Finally, reassign the values in ROW MAJOR order.
	// ************************************************
	x = static_cast<int> (roundf(x_proj));
	y = static_cast<int> (roundf(y_proj));
	
	--x;
	--y;
	
	res_x[idx] = x;
	res_y[idx] = y;
}



namespace kinect {
  
// Projects the depth image onto the RGB image.
// Args:
//   depth_abs_h - depth matrix in row major order. This depth already should have
//               the nonlinearity removed as is measured in meters.
//   depth_proj_h - the projected depth matrix. The result is measured in meters.
//   
void project_depth_par(float* depth_abs_h, float* depth_proj_h) {
	
  int N = 480 * 640;
	
  // Copy the absolute depth values to the device:
	// 
	// depth_abs_h ==> depth_abs_d
	//
	float* depth_abs_d;
  hipMalloc((void**) &depth_abs_d, N * sizeof(float));
  hipMemcpy(depth_abs_d, depth_abs_h, N * sizeof(float), hipMemcpyHostToDevice);
	
	// Allocate the output of the projection.
	int *proj_x_d, *proj_y_d;
	hipMalloc((void**) &proj_x_d, N * sizeof(int));
	hipMalloc((void**) &proj_y_d, N * sizeof(int));
	
  // Execute the parallel projection.
  int block_size = 512;
	int num_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);

  depth_to_point_cloud<<<num_blocks, block_size>>>(depth_abs_d, proj_x_d, proj_y_d, N);
	
  // Copy the result back from the device.
	int *proj_x_h = (int*) malloc(N * sizeof(int));
	int *proj_y_h = (int*) malloc(N * sizeof(int));
	hipMemcpy(proj_x_h, proj_x_d, N * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(proj_y_h, proj_y_d, N * sizeof(int), hipMemcpyDeviceToHost);

  // Create the assignment matrix.
	float* depth_assgn = (float*) malloc(N * sizeof(float));
	for (int nn = 0; nn < N; ++nn) {
	  depth_assgn[nn] = 10;
	}

  // Now, go through the projection results:
	for (int i = 0; i < N; ++i) {
	  int x = proj_x_h[i];
		int y = proj_y_h[i];
		
	  if (x < 0 || x >= WIDTH || y < 0 || y >= HEIGHT) {
	    continue;
	  }

    int abs_offset = y * WIDTH + x;
		
    if (depth_abs_h[i] > depth_assgn[abs_offset]) {
		  continue;
		}

    depth_proj_h[abs_offset] = depth_abs_h[i];
		depth_assgn[abs_offset] = depth_abs_h[i];
	}

	free(proj_x_h);
	free(proj_y_h);
	free(depth_assgn);
	
  // Cleanup.
  hipFree(depth_abs_d);
	hipFree(proj_x_d);
	hipFree(proj_y_d);
}

}  // namespace kinect

